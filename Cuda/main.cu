/* Standard C++ Header Files */

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <ctime>



/* Device Functions Declarations Called By Host */
__global__ void luDecomposition();



/* Device Functions Declarations Called By Device */



/* Host Functions Declarations Called By Host */
__host__ void generateMatrixData(double *, const int *);
__host__ void printMatrix(const double *, const int *);



/* Start Of Program Execution */
int main(int argc, char **argv)
{
  if(argc < 4) { // Cause why not
    fprintf(stderr, "\nERROR: usage: %s <n>\n\t<n>: matrix dimension\n", argv[0]);
    exit(1);
  }

  srand(time(0)); // Seed randomizer

  // Open Cuda Error Output Stream
  FILE *cudaerr = fopen("cuda_error.txt", "w");
  if(cudaerr == NULL) {
    fprintf(cudaerr, "ERROR: Cuda Error File Could Not Be Opened!");
    exit(2);
  }

  // Parse Command Line
  int dimension = atoi(argv[1]);

  // Matrix data
  int n = dimension * dimension;

  // Allocate Host Data
  double *matrix = (double *)calloc(n, sizeof(double));
  double *upperMatrix = (double *)calloc(n, sizeof(double));
  double *lowerMatrix = (double *)calloc(n, sizeof(double));

  // Generate Matrix data 
  generateMatrixData(matrix, &n);
  printMatrix(matrix, &dimension);

  // Allocate Device Data
  double *d_matrix;
  double *d_upperMatrix;
  double *d_lowerMatrix;
  hipMalloc((void **)&d_matrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_upperMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_lowerMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Cuda Stuff
  int block = 256;

  // Start Timer
  std::chrono::time_point<std::chrono::steady_clock> start = std::chrono::steady_clock::now();

  // Copy Host Data To Device Memory
  hipMemcpy(d_matrix, matrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy matrix to d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_upperMatrix, upperMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy upperMatrix to d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_lowerMatrix, lowerMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy lowerMatrix to d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  //luDecomposition<<<(n+block-1)/block,block>>>();
  hipDeviceSynchronize();
  fprintf(cudaerr, "Synchronize after LU Decomposition: %s\n", hipGetErrorString(hipGetLastError()));

  // Copy Device Data to Host Memory
  hipMemcpy(matrix, d_matrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_matrix to matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(upperMatrix, d_upperMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_upperMatrix to upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(lowerMatrix, d_lowerMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_lowerMatrix to lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // End Timer
  std::chrono::time_point<std::chrono::steady_clock> end = std::chrono::steady_clock::now();
  std::chrono::duration<double> time = end - start;

  // Deallocate Device Data
  hipFree(d_matrix);
  fprintf(cudaerr, "Free d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_upperMatrix);
  fprintf(cudaerr, "Free d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_lowerMatrix);
  fprintf(cudaerr, "Free d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Print Time
  fprintf(stdout, "\n%7d\t%14.9f\n", n, time.count());

  // Deallocate Host Data
  free(matrix);
  free(upperMatrix);
  free(lowerMatrix);

  // Close Cuda Error Output Stream
  fclose(cudaerr);

  return 0;
}



/* Host To Device Function Definitions */

__global__ void luDecomposition()
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  //printf("Block ID: %4d | Block Dim: %4d | PE Index:  %4d\n", blockIdx.x, blockDim.x, index);

  return;
}



/* Host To Host Function Definitions */

__host__ void generateMatrixData(double *matrix, const int *n)
{
  for(int i = 0; i < *n; ++i) {
    matrix[i] = rand() % 99 + 1;
  }
 
  return;
}

__host__ void printMatrix(const double *matrix, const int *dimension)
{
  fprintf(stdout, "\nOriginal Matrix:\n");
  for(int i = 0; i < *dimension; ++i) {
    for(int j = 0; j < *dimension; ++j) {
      fprintf(stdout, "%f ", matrix[i*(*dimension)+j]);
    }
    fprintf(stdout, "\n");
  }
  
  return;
}





/* Device To Device Function Definitions */






