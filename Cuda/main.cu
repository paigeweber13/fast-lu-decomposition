/* Standard C++ Header Files */

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <ctime>



/* Device Functions Declarations Called By Host */
__global__ void findTransform(int *, int *, double *, double *);
__global__ void elimination(int *, int *, double *, double *);



/* Device Functions Declarations Called By Device */



/* Host Functions Declarations Called By Host */
__host__ void generateMatrixData(double *, const int *);
__host__ void printMatrix(const double *, const int *);



/* Start Of Program Execution */
int main(int argc, char **argv)
{
  if(argc < 2) { // Cause why not
    fprintf(stderr, "\nERROR: usage: %s <n>\n\t<n>: matrix dimension\n", argv[0]);
    exit(1);
  }

  srand(time(0)); // Seed randomizer

  // Open Cuda Error Output Stream
  FILE *cudaerr = fopen("cuda_error.txt", "w");
  if(cudaerr == NULL) {
    fprintf(cudaerr, "ERROR: Cuda Error File Could Not Be Opened!");
    exit(2);
  }

  // Parse Command Line
  int dimension = 10; //atoi(argv[1]);

  // Matrix data
  int n = dimension * dimension;

  // Allocate Host Data
  double *matrix = (double *)calloc(n, sizeof(double));
  double *upperMatrix = (double *)calloc(n, sizeof(double));
  double *lowerMatrix = (double *)calloc(n, sizeof(double));

  // Generate Matrix data 
  generateMatrixData(matrix, &n);
  printMatrix(matrix, &dimension);


  // Allocate Device Data
  int *d_dimension;
  int *d_n;
  int *d_pivot;
  double *d_matrix;
  double *d_upperMatrix;
  double *d_lowerMatrix;
  hipMalloc((void **)&d_dimension, sizeof(int));
  fprintf(cudaerr, "Allocate d_dimension: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_n, sizeof(int));
  fprintf(cudaerr, "Allocate d_n: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_pivot, sizeof(int));
  fprintf(cudaerr, "Allocate d_pivot: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_matrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_upperMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_lowerMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Start Timer
  std::chrono::time_point<std::chrono::steady_clock> start = std::chrono::steady_clock::now();

  // Copy Host Data To Device Memory
  hipMemcpy(d_matrix, matrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy matrix to d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_upperMatrix, upperMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy upperMatrix to d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_lowerMatrix, lowerMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy lowerMatrix to d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));



  /* Actaul Main Algorithm */

  int pivot;
  for(int i = 0; i < dimension; ++i) {
    pivot = i * dimension + i;
    hipMemcpy(d_pivot, &pivot, sizeof(int), hipMemcpyHostToDevice);
    fprintf(cudaerr, "Copy matrix to d_pivot iteration %d: %s\n", i, hipGetErrorString(hipGetLastError()));

    findTransform<<<1,1>>>(d_pivot, d_dimension, d_matrix, d_lowerMatrix); // Column To 0 And Find Lower

    hipDeviceSynchronize();
    fprintf(cudaerr, "Synchronize after findTransform iteration %d: %s\n", i, hipGetErrorString(hipGetLastError()));

    elimination<<<1,1>>>(d_pivot, d_dimension, d_matrix, d_lowerMatrix);
        
    //luDecomposition<<<(n+block-1)/block,block>>>();
    hipDeviceSynchronize();
    fprintf(cudaerr, "Synchronize after elimination iteration %d: %s\n", i, hipGetErrorString(hipGetLastError()));
  }


  
  // Copy Device Data to Host Memory
  hipMemcpy(matrix, d_matrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_matrix to matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(upperMatrix, d_upperMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_upperMatrix to upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(lowerMatrix, d_lowerMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_lowerMatrix to lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipDeviceSynchronize();
  fprintf(cudaerr, "Synchronize after copying to device: %s\n", hipGetErrorString(hipGetLastError()));

  // End Timer
  std::chrono::time_point<std::chrono::steady_clock> end = std::chrono::steady_clock::now();
  std::chrono::duration<double> time = end - start;

  // Deallocate Device Data
  hipFree(d_dimension);
  fprintf(cudaerr, "Free d_dimension: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_n);
  fprintf(cudaerr, "Free d_n: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_pivot);
  fprintf(cudaerr, "Free d_pivot: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_matrix);
  fprintf(cudaerr, "Free d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_upperMatrix);
  fprintf(cudaerr, "Free d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_lowerMatrix);
  fprintf(cudaerr, "Free d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

printMatrix(matrix, &dimension);
printMatrix(upperMatrix, &dimension);
printMatrix(lowerMatrix, &dimension);

  // Print Time
  fprintf(stdout, "\n%7d\t%14.9f\n", n, time.count());

  // Deallocate Host Data
  free(matrix);
  free(upperMatrix);
  free(lowerMatrix);

  // Close Cuda Error Output Stream
  fclose(cudaerr);

  return 0;
}



/* Host To Device Function Definitions */

__global__ void findTransform(int *pivot, int *dimension, double *matrix, double *lowerMatrix)
{ 
  double factor = matrix[(*pivot) * (*dimension) + (*pivot)];
  for(int i = (*pivot); i < (*dimension); ++i) {
    lowerMatrix[i*(*dimension)] = matrix[i*(*dimension)] / factor;
  }
}


__global__ void elimination(int *pivot, int *dimension, double *matrix, double *lowerMatrix)
{
  //int index = threadIdx.x + blockIdx.x * blockDim.x;
  for(int i = (*pivot) + 1; i < (*dimension); ++i) {
    matrix[i+(*pivot)*(*dimension)] = lowerMatrix[i*(*dimension)] * matrix[i+(*pivot)*(*dimension)];
  }

  return;
}



/* Host To Host Function Definitions */

__host__ void generateMatrixData(double *matrix, const int *n)
{
  for(int i = 0; i < *n; ++i) {
    matrix[i] = rand() % 99 + 1;
  }
 
  return;
}


__host__ void printMatrix(const double *matrix, const int *dimension)
{
  fprintf(stdout, "\nOriginal Matrix:\n");
  for(int i = 0; i < *dimension; ++i) {
    for(int j = 0; j < *dimension; ++j) {
      fprintf(stdout, "%8.4f ", matrix[i*(*dimension)+j]);
    }
    fprintf(stdout, "\n");
  }
  
  return;
}





/* Device To Device Function Definitions */






