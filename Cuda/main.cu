/* Standard C++ Header Files */

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <ctime>

__global__ void findTransform(int *, int *, double *, double *);
__global__ void elimination(int *, int *, double *, double *);
__host__ void generateMatrixData(double *, int *);
__host__ void printMatrix(double *, int *);


/* Start Of Program Execution */
int main(int argc, char **argv)
{
  if(argc < 2) { // Cause why not
    fprintf(stderr, "\nERROR: usage: %s <n>\n\t<n>: matrix dimension\n", argv[0]);
    exit(1);
  }

  srand(time(0)); // Seed randomizer

  FILE *cudaerr = fopen("cuda_error.txt", "w");
  if(cudaerr == NULL) {
    fprintf(cudaerr, "ERROR: Cuda Error File Could Not Be Opened!");
    exit(2);
  }

  // Parse Command Line (edit once case 10 works)
  int dimension = 10; //atoi(argv[1]);

  // Matrix data
  int n = dimension * dimension;

  // Allocate Host Data
  double *matrix = (double *)calloc(n, sizeof(double));
  double *upperMatrix = (double *)calloc(n, sizeof(double));
  double *lowerMatrix = (double *)calloc(n, sizeof(double));

  // Generate Matrix data 
  generateMatrixData(matrix, &n);
  printMatrix(matrix, &dimension);

  // Allocate Device Data
  int *d_dimension;
  int *d_n;
  int *d_pivot;
  double *d_matrix;
  double *d_upperMatrix;
  double *d_lowerMatrix;
  hipMalloc((void **)&d_dimension, sizeof(int));
  fprintf(cudaerr, "Allocate d_dimension: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_n, sizeof(int));
  fprintf(cudaerr, "Allocate d_n: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_pivot, sizeof(int));
  fprintf(cudaerr, "Allocate d_pivot: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_matrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_upperMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_lowerMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Start Timer
  std::chrono::time_point<std::chrono::steady_clock> start = std::chrono::steady_clock::now();

  // Copy Host Data To Device Memory
  hipMemcpy(d_dimension, &dimension, sizeof(int), hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy lowerMatrix to dimension: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_matrix, matrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy matrix to d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_upperMatrix, upperMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy upperMatrix to d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_lowerMatrix, lowerMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy lowerMatrix to d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));


  /* Actaul Main Algorithm */

  for(int i = 0; i < dimension; ++i) { // Loop through pivots
    int pivot = i * dimension + i;
    hipMemcpy(d_pivot, &pivot, sizeof(int), hipMemcpyHostToDevice);
    fprintf(cudaerr, "Copy matrix to d_pivot iteration %d: %s\n", i, hipGetErrorString(hipGetLastError())); // Remove for benchmarking

    findTransform<<<1,1>>>(d_pivot, d_dimension, d_matrix, d_lowerMatrix); // Column To 0 And Find Lower (can this be done in parallel)?

    hipDeviceSynchronize(); // Do wee need this?
    fprintf(cudaerr, "Synchronize after findTransform iteration %d: %s\n", i, hipGetErrorString(hipGetLastError())); // Remove for benchmarking

    elimination<<<1,1>>>(d_pivot, d_dimension, d_matrix, d_lowerMatrix); // To be done in parallel
        
    hipDeviceSynchronize(); // Also do we need this?
    fprintf(cudaerr, "Synchronize after elimination iteration %d: %s\n", i, hipGetErrorString(hipGetLastError())); // Remove for benchmarking
  }

  // Copy Device Data to Host Memory
  hipMemcpy(matrix, d_matrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_matrix to matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(upperMatrix, d_upperMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_upperMatrix to upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(lowerMatrix, d_lowerMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_lowerMatrix to lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipDeviceSynchronize();
  fprintf(cudaerr, "Synchronize after copying to device: %s\n", hipGetErrorString(hipGetLastError()));

  // End Timer
  std::chrono::time_point<std::chrono::steady_clock> end = std::chrono::steady_clock::now();
  std::chrono::duration<double> time = end - start;

  // Printing for testing
  printMatrix(matrix, &dimension);
  printMatrix(upperMatrix, &dimension);
  printMatrix(lowerMatrix, &dimension);

  // Deallocate Device Data
  hipFree(d_dimension);
  fprintf(cudaerr, "Free d_dimension: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_n);
  fprintf(cudaerr, "Free d_n: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_pivot);
  fprintf(cudaerr, "Free d_pivot: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_matrix);
  fprintf(cudaerr, "Free d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_upperMatrix);
  fprintf(cudaerr, "Free d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_lowerMatrix);
  fprintf(cudaerr, "Free d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Print Time
  fprintf(stdout, "\n%7d\t%14.9f\n", n, time.count());

  // Deallocate Host Data
  free(matrix);
  free(upperMatrix);
  free(lowerMatrix);

  // Close Cuda Error Output Stream
  fclose(cudaerr);

  return 0;
}


__global__ void findTransform(int *pivot, int *dimension, double *matrix, double *lowerMatrix)
{ 
  double factor = matrix[(*pivot) * (*dimension) + (*pivot)];
  for(int i = (*pivot), j = (*pivot); i < (*dimension)*(*dimension); i +=(*dimension), ++j) {
    lowerMatrix[(*pivot)+j*(*dimension)] = matrix[j+(*pivot)*(*dimension)] / factor;
  }
}


__global__ void elimination(int *pivot, int *dimension, double *matrix, double *lowerMatrix)
{
  //int index = threadIdx.x + blockIdx.x * blockDim.x;
  for(int j = (*pivot); j < (*dimension); ++j) {
    for(int i = (*pivot); i < (*dimension); ++i) {
      matrix[(i+(*pivot + 1)*(*dimension))+j*(*dimension)] = (-1) * lowerMatrix[j*(*dimension)] * matrix[j+(*pivot)*(*dimension)] + matrix[i+j*(*dimension)];
    }
  }
  return;
}


__host__ void generateMatrixData(double *matrix, int *n)
{
  for(int i = 0; i < *n; ++i) {
    matrix[i] = rand() % 99 + 1;
  }
 
  return;
}


__host__ void printMatrix(double *matrix, int *dimension)
{
  fprintf(stdout, "\nMatrix:\n");
  for(int i = 0; i < *dimension; ++i) {
    for(int j = 0; j < *dimension; ++j) {
      fprintf(stdout, "%8.4f ", matrix[i*(*dimension)+j]);
    }
    fprintf(stdout, "\n");
  }
  
  return;
}
