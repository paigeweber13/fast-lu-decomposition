#include "hip/hip_runtime.h"
/* Standard C++ Header Files */
#include <chrono>
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <ctime>



/* Device Functions Declarations Called By Host */
__global__ void luDecomposition();



/* Device Functions Declarations Called By Device */



/* Host Functions Declarations Called By Host */
__host__ void generateMatrixData(const double *, const int32_t *);
__host__ void printMatrix(const double *, const int32_t *);



/* Start Of Program Execution */
int main(int argc, char **argv)
{
  if(argc < 4) { // Cause why not
    fprintf(stderr, "\nERROR: usage: %s <n>\n\t<n>: matrix dimension\n", argv[0]);
    exit(1);
  }

  srand(time(0)); // Seed randomizer

  // Open Cuda Error Output Stream
  FILE *cudaerr = fopen("cuda_error.txt", "w");
  if(cudaerr == NULL) {
    fprintf(cudaerr, "ERROR: Cuda Error File Could Not Be Opened!");
    exit(2);
  }

  // Parse Command Line
  int32_t dimension = atoi(argv[1]);

  // Matrix data
  n = dimension * dimension;

  // Allocate Host Data
  double *matrix = calloc(n, sizeof(double));
  double *upperMatrix = calloc(n, sizeof(double));
  double *lowerMatrix = calloc(n, sizeof(double));

  // Generate Matrix data 
  generateMatrixData(matrix, &n);
  #if DEBUG == 1
    printMatrix(matrix, dimension);
  #endif

  // Allocate Device Data
  double *d_matrix;
  double *d_upperMatrix;
  double *d_lowerMatrix;
  hipMalloc((void **)&d_matrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_upperMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_lowerMatrix, sizeof(double) * n);
  fprintf(cudaerr, "Allocate d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Cuda Stuff
  int32_t block = 256;

  // Start Timer
  std::chrono::time_point<std::chrono::steady_clock> start = std::chrono::steady_clock::now();

  // Copy Host Data To Device Memory
  hipMemcpy(d_matrix, matrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy matrix to d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_upperMatrix, upperMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy upperMatrix to d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_lowerMatrix, lowerMatrix, sizeof(double) * n, hipMemcpyHostToDevice);
  fprintf(cudaerr, "Copy lowerMatrix to d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  luDecomposition<<<(n+block-1)/block,block>>>(d_array, d_poly, d_n, d_degree);
  hipDeviceSynchronize();
  fprintf(cudaerr, "Synchronize after LU Decomposition: %s\n", hipGetErrorString(hipGetLastError()));

  // Copy Device Data to Host Memory
  hipMemcpy(matrix, d_matrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_matrix to matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(upperMatrix, d_upperMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_upperMatrix to upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(lowerMatrix, d_lowerMatrix, sizeof(double) * n, hipMemcpyDeviceToHost);
  fprintf(cudaerr, "Copy d_lowerMatrix to lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // End Timer
  std::chrono::time_point<std::chrono::steady_clock> end = std::chrono::steady_clock::now();
  std::chrono::duration<double> time = end - start;

  // Deallocate Device Data
  hipFree(d_matrix);
  fprintf(cudaerr, "Free d_matrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_upperMatrix);
  fprintf(cudaerr, "Free d_upperMatrix: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_lowerMatrix);
  fprintf(cudaerr, "Free d_lowerMatrix: %s\n", hipGetErrorString(hipGetLastError()));

  // Print Time
  fprintf(stdout, "\n%7d\t%14.9f\n", n, time.count());

  // Deallocate Host Data
  free(matrix);
  free(uppermatrix);
  free(lowerMatrix);

  // Close Cuda Error Output Stream
  fclose(cudaerr);

  return 0;
}



/* Host To Device Function Definitions */

__global__ void polynomialExpansion(float *array, const float *poly, const int32_t *n, const int32_t *degree)
{
  int32_t index = threadIdx.x + blockIdx.x * blockDim.x;
  //printf("Block ID: %4d | Block Dim: %4d | PE Index:  %4d\n", blockIdx.x, blockDim.x, index);
  if(index < *n) {
    //printf("array: %f\n", array[index]);
    array[index] = polynomial(&array[index], &poly[index], degree);
    //printf("array: %f\n", array[index]);
  }

  return;
}



/* Host To Host Function Definitions */

__host__ generateMatrixData(const double *matrix, const int32_t *n)
{
  for(int32_t i = 0; i < n; ++i) {
    #if DEBUG == 1
      matrix[i] = rand() % 99 + 1;
    #else
      matrix[i] = rand() % 999999 + 1;
    #endif
  }
 
  return;
}

__host__ printMatrix(const double *matrix, const int32_t *dimension)
{
  fprintf(stdout, "\nOriginal Matrix:\n");
  for(int32_t i = 0; i < dimension; ++i) {
    for(int32_t j = 0; j < dimension; ++j) {
      fprintf(stdout, "%f ", matrix[i*dimension+j]);
    }
    fprintf(stdout, "\n"_'
  }
  
  return;
}





/* Device To Device Function Definitions */






